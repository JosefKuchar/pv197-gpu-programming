#include "hip/hip_runtime.h"
// TODO: Use advanced techniques from
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

const int BLOCK_SIZE = 512;

__device__ void warpReduce(volatile int* data, int threadId) {
    data[threadId] += data[threadId + 32];
    data[threadId] += data[threadId + 16];
    data[threadId] += data[threadId + 8];
    data[threadId] += data[threadId + 4];
    data[threadId] += data[threadId + 2];
    data[threadId] += data[threadId + 1];
}

__global__ void sumScanSingleBlock(int* changes, int* account, int clients) {
    int clientId = blockIdx.x;
    int threadId = threadIdx.x;
    int periods = blockDim.x * 2;

    int shared1Id = threadIdx.x * 2;
    int shared2Id = threadIdx.x * 2 + 1;

    int global1Id = clients * (shared1Id + blockDim.x * blockIdx.y * 2) + clientId;
    int global2Id = clients * (shared2Id + blockDim.x * blockIdx.y * 2) + clientId;

    extern __shared__ int shared[];
    // Load data into shared memory
    shared[shared1Id] = changes[global1Id];
    shared[shared2Id] = changes[global2Id];

    // Reduction phase
    int offset = 1;
    for (int d = periods >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            shared[bi] += shared[ai];
        }
        offset *= 2;
    }

    // Clear the last element
    if (threadId == 0) {
        shared[periods - 1] = 0;
    }

    // Post-reduction phase
    for (int d = 1; d < periods; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            int t = shared[ai];
            shared[ai] = shared[bi];
            shared[bi] += t;
        }
    }

    __syncthreads();
    // Write results to global memory
    shared[shared1Id] += changes[global1Id];
    shared[shared2Id] += changes[global2Id];
    account[global1Id] = shared[shared1Id];
    account[global2Id] = shared[shared2Id];
}

__global__ void sumScanMultiBlock(int* changes, int* account, int clients, int* temp) {
    int clientId = blockIdx.x;
    int threadId = threadIdx.x;
    int periods = blockDim.x * 2;

    int shared1Id = threadIdx.x * 2;
    int shared2Id = threadIdx.x * 2 + 1;

    int global1Id = clients * (shared1Id + blockDim.x * blockIdx.y * 2) + clientId;
    int global2Id = clients * (shared2Id + blockDim.x * blockIdx.y * 2) + clientId;

    // Block dim x - number of clients
    // Block dim y - 2

    extern __shared__ int shared[];
    // Load data into shared memory
    shared[shared1Id] = changes[global1Id];
    shared[shared2Id] = changes[global2Id];
    shared[shared1Id + blockDim.x * 2] = shared[shared1Id];
    shared[shared2Id + blockDim.x * 2] = shared[shared2Id];

    // Reduction sum
    for (int i = 1; i < blockDim.x * 2; i *= 2) {
        if (threadId % i == 0) {
            shared[threadId * 2 + blockDim.x * 2] += shared[threadId * 2 + i + blockDim.x * 2];
        }
        __syncthreads();
    }

    if (threadId == 0) {
        for (int i = blockIdx.y + 1; i < gridDim.y; i++) {
            atomicAdd(&temp[gridDim.y * blockIdx.x + i], shared[blockDim.x * 2]);
        }
        // atomicExch(&atom[gridDim.y * blockIdx.x + blockIdx.y], 1);
    }

    // Reduction phase
    int offset = 1;
    for (int d = periods >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            shared[bi] += shared[ai];
        }
        offset *= 2;
    }

    // Clear the last element
    if (threadId == 0) {
        shared[periods - 1] = 0;
    }

    // Post-reduction phase
    for (int d = 1; d < periods; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (threadId < d) {
            int ai = offset * (2 * threadId + 1) - 1;
            int bi = offset * (2 * threadId + 2) - 1;
            int t = shared[ai];
            shared[ai] = shared[bi];
            shared[bi] += t;
        }
    }

    __syncthreads();
    // Write results to global memory
    shared[shared1Id] += changes[global1Id] + temp[gridDim.y * blockIdx.x + blockIdx.y];
    shared[shared2Id] += changes[global2Id] + temp[gridDim.y * blockIdx.x + blockIdx.y];
    account[global1Id] = shared[shared1Id];
    account[global2Id] = shared[shared2Id];
}

__global__ void sumReduceSingleBlock(int* account, int* sum, int clients, int periods) {
    extern __shared__ int shared[];
    // Each thread loads two elements into shared memory
    shared[threadIdx.x] = account[blockIdx.x * clients + threadIdx.x];
    __syncthreads();

    for (int i = blockDim.x / 2; i > 32; i >>= 1) {
        if (threadIdx.x < i) {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
        __syncthreads();
    }
    if (threadIdx.x < 32) {
        warpReduce(shared, threadIdx.x);
    }
    if (threadIdx.x == 0) {
        sum[blockIdx.x] = shared[0];
    }
}

// __global__ void sumReduceSingleBlock(int* account, int* sum, int clients, int periods) {
//     extern __shared__ int shared[];
//     // Each thread loads two elements into shared memory
//     shared[threadIdx.x * 2] =
//         account[blockIdx.x * clients + threadIdx.x * 2 + blockIdx.y * blockDim.x * 2];
//     shared[threadIdx.x * 2 + 1] =
//         account[blockIdx.x * clients + threadIdx.x * 2 + 1 + blockIdx.y * blockDim.x * 2];

//     for (int i = 1; i < blockDim.x * 2; i *= 2) {
//         if (threadIdx.x % i == 0) {
//             shared[threadIdx.x * 2] += shared[threadIdx.x * 2 + i];
//         }
//         __syncthreads();
//     }
//     if (threadIdx.x == 0) {
//         sum[blockIdx.x] = shared[0];
//     }
// }

// __global__ void kernel3(int* changes, int* account, int* sum, int clients, int* temp) {
//     int globalId = clients * (threadIdx.x + blockDim.x * blockIdx.y) + blockIdx.x;
//     account[globalId] += temp[gridDim.y * blockIdx.x + blockIdx.y];
// }

__global__ void sumkernel(int* account, int* sum, int clients, int periods, int* temp) {
    extern __shared__ int shared[];
    // Each thread loads two elements into shared memory
    shared[threadIdx.x * 2] =
        account[blockIdx.x * clients + threadIdx.x * 2 + blockIdx.y * blockDim.x * 2];
    shared[threadIdx.x * 2 + 1] =
        account[blockIdx.x * clients + threadIdx.x * 2 + 1 + blockIdx.y * blockDim.x * 2];

    for (int i = 1; i < blockDim.x * 2; i *= 2) {
        if (threadIdx.x % i == 0) {
            shared[threadIdx.x * 2] += shared[threadIdx.x * 2 + i];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        temp[blockIdx.x * gridDim.y + blockIdx.y] = shared[0];
    }
}

__global__ void sumkernel2(int* account, int* sum, int clients, int periods, int* temp) {
    extern __shared__ int shared[];
    shared[threadIdx.x * 2] = temp[blockIdx.x * 8 + threadIdx.x * 2];
    shared[threadIdx.x * 2 + 1] = temp[blockIdx.x * 8 + threadIdx.x * 2 + 1];

    for (int i = 1; i < blockDim.x * 2; i *= 2) {
        if (threadIdx.x % i == 0) {
            shared[threadIdx.x * 2] += shared[threadIdx.x * 2 + i];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        sum[blockIdx.x] = shared[0];
    }
}

void solveGPU(int* changes, int* account, int* sum, int clients, int periods) {
    //int* temp;
    //hipMalloc((void**)&temp, sizeof(int) * clients * (periods / BLOCK_SIZE) / 2);
    /**
     * If periods is less than 2048, we can use kernel that calculates everything in one block
     * Otherwise we need to synchronize multiple blocks and that creates a lot of overhead
     */
    /*
    if (periods <= 2048) {
        int memory = sizeof(int) * periods;
        sumScanSingleBlock<<<clients, periods / 2, memory>>>(changes, account, clients);
    } else {
        int memory = sizeof(int) * (BLOCK_SIZE * 4);
        dim3 grid(clients, (periods / BLOCK_SIZE) / 2);
        sumScanMultiBlock<<<grid, BLOCK_SIZE, memory>>>(changes, account, clients, temp);
    }

    {
        int memory = sizeof(int) * clients;
        sumReduceSingleBlock<<<periods, clients, memory>>>(account, sum, clients, periods);
    }*/
    // dim3 grid(clients, (periods / BLOCK_SIZE) / 2);
    // dim3 block(BLOCK_SIZE);
    // kernel2<<<grid, block, sizeof(int) * (BLOCK_SIZE * 4)>>>(changes, account, sum, clients,
    // temp); kernel3<<<grid, BLOCK_SIZE * 2>>>(changes, account, sum, clients, temp);
    // dim3 sumblocks(periods, (clients / BLOCK_SIZE) / 2);
    // sumkernel<<<sumblocks, BLOCK_SIZE, sizeof(int) * (BLOCK_SIZE * 2)>>>(account, sum, clients,
    //                                                                      periods, temp);
    // int threads = (clients / BLOCK_SIZE) / 4;
    // // printf("Threads: %d\n", threads);
    // sumkernel2<<<periods, threads, sizeof(int) * threads * 2>>>(account, sum, clients, periods,
    //                                                             temp);
    kernel<<<clients, 1>>>(changes, account, sum, clients, periods);
}
