
#include <hip/hip_runtime.h>
// TODO: Use advanced techniques from
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// https://web.archive.org/web/20110924131401/http://www.moderngpu.com/intro/scan.html

#define COLS 8
#define ROWS 4
#define BLOCK_ROWS 64

// __global__ void firstStage(int* changes, int* account, int* sum, int clients, int periods) {
//     __shared__ volatile int temp[BLOCK_COLS * ROWS * 2];

//     int tx = threadIdx.x % COLS;
//     int ty = (threadIdx.x / COLS) % ROWS;
//     int xx = threadIdx.x / (COLS * ROWS);

//     int index = tx + xx * COLS + blockIdx.x * BLOCK_COLS + clients * ty;

//     int sharedIndex = ty + tx * (ROWS + ROWS / 2) + xx * (ROWS * COLS * 2);
//     int prev = 0;
//     for (int i = 0; i < 2048; i++) {
//         temp[sharedIndex] = changes[index] + prev;
//         temp[sharedIndex + 1] += temp[sharedIndex];
//         temp[sharedIndex + 2] += temp[sharedIndex];
//         if (ty == 0) {
//             prev = temp[sharedIndex + 3];
//         }
//         account[index] = temp[sharedIndex];
//         // atomicAdd(&sum[4 * i + ty], temp[sharedIndex]);
//         index += clients * ROWS;
//     }
// }

__global__ void kernel(int* changes, int* account, int* sum, int clients, int periods) {
    __shared__ volatile int shared[COLS * BLOCK_ROWS];
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int index = tx + blockIdx.x * COLS + clients * ty;
    int sharedIndex = tx + ty * COLS;

    for (int i = 0; i < 128; i++) {
        changes[index] = account[index];
        index += clients * BLOCK_ROWS;
    }

    // int tx = threadIdx.x % COLS;
    // int ty = (threadIdx.x / COLS) % ROWS;
    // int xx = threadIdx.x / (COLS * ROWS);

    // int index = tx + xx * COLS + blockIdx.x * BLOCK_COLS + clients * ty;

    // int sharedIndex = ty + tx * ROWS + xx * (ROWS * COLS);
    // int prev = 0;
    // for (int i = 0; i < 2048; i++) {
    //     temp[sharedIndex] = changes[index] + prev;
    //     if (ty < 3) {
    //         temp[sharedIndex + 1] += temp[sharedIndex];
    //     }
    //     if (ty < 2) {
    //         temp[sharedIndex + 2] += temp[sharedIndex];
    //     }
    //     if (ty == 0) {
    //         prev = temp[sharedIndex + 3];
    //     }
    //     account[index] = temp[sharedIndex];
    //     if (tx < 4) {
    //         temp[sharedIndex] += temp[sharedIndex + 4 * 4];
    //         temp[sharedIndex] += temp[sharedIndex + 4 * 2];
    //         temp[sharedIndex] += temp[sharedIndex + 4];
    //     }
    //     if (tx == 0) {
    //         atomicAdd(&sum[4 * i + ty], temp[sharedIndex]);
    //     }
    //     index += clients * ROWS;
    // }
}

void solveGPU(int* changes, int* account, int* sum, int clients, int periods) {
    dim3 block(COLS, BLOCK_ROWS);
    kernel<<<clients / COLS, block>>>(changes, account, sum, clients, periods);

    // Output memory errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}
