
#include <hip/hip_runtime.h>
// TODO: Use advanced techniques from
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// https://web.archive.org/web/20110924131401/http://www.moderngpu.com/intro/scan.html

#define TILE 16

// __global__ void firstStage(int* changes, int* account, int* sum, int clients, int periods) {
//     __shared__ volatile int temp[BLOCK_COLS * ROWS * 2];

//     int tx = threadIdx.x % COLS;
//     int ty = (threadIdx.x / COLS) % ROWS;
//     int xx = threadIdx.x / (COLS * ROWS);

//     int index = tx + xx * COLS + blockIdx.x * BLOCK_COLS + clients * ty;

//     int sharedIndex = ty + tx * (ROWS + ROWS / 2) + xx * (ROWS * COLS * 2);
//     int prev = 0;
//     for (int i = 0; i < 2048; i++) {
//         temp[sharedIndex] = changes[index] + prev;
//         temp[sharedIndex + 1] += temp[sharedIndex];
//         temp[sharedIndex + 2] += temp[sharedIndex];
//         if (ty == 0) {
//             prev = temp[sharedIndex + 3];
//         }
//         account[index] = temp[sharedIndex];
//         // atomicAdd(&sum[4 * i + ty], temp[sharedIndex]);
//         index += clients * ROWS;
//     }
// }

__global__ void kernel(int* changes, int* account, int* sum, int clients, int periods) {
    __shared__ volatile int shared[TILE * (TILE + 1)];

    // int tx = threadIdx.x % COLS;
    // int ty = (threadIdx.x / COLS) % ROWS;
    // int xx = threadIdx.x / (COLS * ROWS);

    int index = threadIdx.x + blockIdx.x * TILE + clients * threadIdx.y;
    int si = threadIdx.x + threadIdx.y * TILE;
    for (int i = 0; i < 512; i++) {
        shared[si] = changes[index];
        __syncthreads();
        account[index] = shared[si];
        index += clients * TILE;
    }
}

void solveGPU(int* changes, int* account, int* sum, int clients, int periods) {
    dim3 block(TILE, TILE);
    kernel<<<clients / TILE, block>>>(changes, account, sum, clients, periods);

    // Output memory errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}
