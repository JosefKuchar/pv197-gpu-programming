
#include <hip/hip_runtime.h>
// TODO: Use advanced techniques from
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
// https://web.archive.org/web/20110924131401/http://www.moderngpu.com/intro/scan.html

#define BLOCK_COLS 32
#define COLS 8
#define ROWS 4

__global__ void firstStage(int* changes, int* account, int* sum, int clients, int periods) {
    __shared__ volatile int temp[BLOCK_COLS * ROWS];

    int tx = threadIdx.x % COLS;
    int ty = (threadIdx.x / COLS) % ROWS;
    int xx = threadIdx.x / (COLS * ROWS);

    int index = tx + xx * COLS + blockIdx.x * BLOCK_COLS + clients * ty;

    int sharedIndex = ty + tx * ROWS + xx * (ROWS * COLS);
    for (int i = 0; i < 2048; i++) {
        temp[sharedIndex] = changes[index];

        int v = temp[sharedIndex];
        if (ty < 3) {
            temp[sharedIndex + 1] += v;
        }
        account[index] = temp[sharedIndex];
        index += clients * ROWS;
    }
}

void solveGPU(int* changes, int* account, int* sum, int clients, int periods) {
    firstStage<<<clients / BLOCK_COLS, BLOCK_COLS * ROWS>>>(changes, account, sum, clients,
                                                            periods);

    // Output memory errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
}
