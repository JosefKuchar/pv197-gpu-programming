
#include <hip/hip_runtime.h>
// TODO: Use advanced techniques from
// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda

__global__ void kernel(int* changes, int* account, int* sum, int clients, int periods) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    account[i] = changes[i];
    atomicAdd(&sum[0], account[i]);
    for (int j = 1; j < periods; j++) {
        account[j * clients + i] = account[(j - 1) * clients + i] + changes[j * clients + i];
        atomicAdd(&sum[j], account[j * clients + i]);
    }
}

void solveGPU(int* changes, int* account, int* sum, int clients, int periods) {
    kernel<<<clients, 1>>>(changes, account, sum, clients, periods);
}
